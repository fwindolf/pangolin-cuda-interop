#include "hip/hip_runtime.h"

#include "kernel.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void kernel(hipTextureObject_t tex, T* output, int width, int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;
    
    unsigned int x_new = (x + 3) % width;
    
    T data = tex2D<T>(tex, x_new, y);
    output[x + y * width] = data;
}

template __global__ void kernel(hipTextureObject_t tex, uchar1* output, int width, int height);
template __global__ void kernel(hipTextureObject_t tex, uchar4* output, int width, int height);
template __global__ void kernel(hipTextureObject_t tex, float1* output, int width, int height);
template __global__ void kernel(hipTextureObject_t tex, float4* output, int width, int height);

template<typename T>
void transformImage(hipArray_t input, T* output, int width, int height)
{
    hipTextureObject_t tex;

    hipResourceDesc texRes;
    memset(&texRes,0, sizeof(hipResourceDesc));
    texRes.resType            = hipResourceTypeArray;
    texRes.res.array.array    = input;

    hipTextureDesc texDescr;
    memset(&texDescr,0,sizeof(hipTextureDesc));
    // texDescr.normalizedCoords = 1;
    // texDescr.filterMode       = hipFilterModeLinear;
    texDescr.addressMode[0]   = hipAddressModeClamp;
    texDescr.addressMode[1]   = hipAddressModeClamp;
    texDescr.readMode         = hipReadModeElementType;
    
    cudaSafeCall(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
    
    // Invoke kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);

    kernel <<<dimGrid, dimBlock>>> (tex, (T*)output, width, height);    
    cudaCheck(hipGetLastError(), __FILE__, __LINE__);

    cudaSafeCall(hipDestroyTextureObject(tex));
}


template void transformImage(hipArray_t, uchar1*, int, int);
template void transformImage(hipArray_t, uchar4*, int, int);
template void transformImage(hipArray_t, float1*, int, int);
template void transformImage(hipArray_t, float4*, int, int);
